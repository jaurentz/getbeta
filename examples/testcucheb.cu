#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cucheb.h>
#include <omp.h>


int main(void){

	//defining variables
	int n;
	double *x, *y, *dx, *dy, *A, *C, *P;
	
	// set size
	n = 200;
	
	// allocate host memory
	x = new double[n];
	y = new double[n];
	
	// print values in host memory
	//printf("\nx and y before being set:\n\n");
	//for(int ii=0;ii<n;ii++){
	//	printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	//}
	//printf("\n");
	
	// allocate device memory
	cuchebCheckError(hipMalloc(&dx,n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dy,n*sizeof(double)),__FILE__,__LINE__);

	cuchebCheckError(hipMalloc(&A,n*n*sizeof(double)),__FILE__,__LINE__); 		
	cuchebCheckError(hipMalloc(&C,n*n*sizeof(double)),__FILE__,__LINE__); 	
	cuchebCheckError(hipMalloc(&P,n*n*sizeof(double)),__FILE__,__LINE__); 	

	// initialize hiprand
	hiprandGenerator_t curand_gen;
	cuchebCheckError(hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT),__FILE__,__LINE__);
	cuchebCheckError(hiprandSetPseudoRandomGeneratorSeed(curand_gen,time(NULL)),__FILE__,__LINE__);
	
	// random starting vector 'A'.
	cuchebCheckError(hiprandGenerateNormalDouble(curand_gen,A,n*n,0.0,1.0),__FILE__,__LINE__);

	// set device memory using CUCHEB
	cuchebCheckError(cuchebDinit(n,dx,1,1.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n,dy,1,1.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n*n,C,1,0.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n*n,P,1,0.0),__FILE__,__LINE__);	
	cuchebCheckError(cuchebDinit(n,P,n+1,1.0),__FILE__,__LINE__); 
	
	//cuchebCheckError(cuchebDinit(n*n,A,1,cuRAND),__FILE__,__LINE__);
	
	//this sets the identity matrix 'A'	
	//cuchebCheckError(cuchebDinit(n,A,n+1,1.0),__FILE__,__LINE__); 

<<<<<<< Updated upstream
	//cuchebCheckError(cuchebDinit(n*n,A,1,cuRAND),__FILE__,__LINE__);
	//cuchebCheckError(cuchebDinit(n,A,n+1,1.0),__FILE__,__LINE__);
=======
>>>>>>> Stashed changes
	

	// create cublas handle
	hipblasHandle_t handle;
	cuchebCheckError(hipblasCreate(&handle),__FILE__,__LINE__);
	
	// set pointer mode to host
	cuchebCheckError(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);

	// compute norm of dx
	//double nrmx;
	//cuchebCheckError(hipblasDnrm2(handle, n, dx, 1, &nrmx),__FILE__,__LINE__);

	// compute sum of dx
	//double sum;
	//cuchebCheckError(hipblasDasum(handle, n, dx, 1, &sum),__FILE__,__LINE__);

	// print sum of x
	//printf("\nthe sum of x = %E \n\n", sum);


	//computing alpha times a vector(dx) and storing it in (dy)
	//double alpha = 2.0;
	//int incx = 1; 
	//cuchebCheckError( hipblasDaxpy(handle, n, &alpha, dx, incx, dy, incx),__FILE__,__LINE__);


	//compute the dot product between the two vectors 
	//double z;
	//int incx = 1;
	//cuchebCheckError(hipblasDdot (handle, n, dx, incx, dy, incx, &z),__FILE__,__LINE__);

	//print the dot product "result"
	//printf("\n this is the dot product between x and y: \n %E", z); 


	//function performs matrix-vector multiplication y = alpha * OP(A) * x + beta * y
	//int stride = 1;		
	//double alpha = 2.0;
	//double beta = 3.0;
	//cuchebCheckError(hipblasDgemv(handle, HIPBLAS_OP_T, n, n, &alpha, A, n, dx, stride, &beta, dy, stride),__FILE__,__LINE__);	
	
	// compute norm of dy
	//double nrmy;
	//cuchebCheckError(hipblasDnrm2(handle, n, dy, 1, &nrmy),__FILE__,__LINE__);

	// print norm of y
	//printf("\nnorm of y = %+1.15e\n\n",nrmy);


	//attempt at gram-schit process.
	double numerator, denominator, norm, alpha; 
	int ii, jj; 
	int stride = 1;
	for(ii=0;ii<n;ii++){
		//pick the sequential 'n' elements of 'A'.
			for(jj=0;jj<ii;jj++){
	
			cuchebCheckError(hipblasDdot (handle, n, &A[ii*n], stride, &A[jj*n], stride, &numerator),__FILE__,__LINE__) ;

			cuchebCheckError(hipblasDdot(handle, n, &A[jj*n], stride, &A[jj*n], stride, &denominator),__FILE__,__LINE__);
			
			alpha = -numerator / denominator; 

			cuchebCheckError(hipblasDaxpy(handle, n, &alpha, &A[jj*n], stride, &A[ii*n], stride),__FILE__,__LINE__);

		}  
	//computing the L2 norm from the orthogonal-ized vector from above 
	cuchebCheckError(hipblasDnrm2(handle, n, &A[ii*n], stride, &norm),__FILE__,__LINE__);

	norm = 1.0/norm ; 

	cuchebCheckError(hipblasDscal(handle, n, &norm, &A[ii*n], stride),__FILE__,__LINE__);

	}	

	//checking the gram-schit algorithm... 
	alpha = 1.0;
	cuchebCheckError(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n, &alpha, A, n, A, n, &alpha, C, n),__FILE__,__LINE__);

	double beta = -1.0;
	//double Z; 
	cuchebCheckError(hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, C, n, &beta, P, n, P, n),__FILE__,__LINE__);

	double check_norm;
	cuchebCheckError(hipblasDnrm2(handle, n*n, P, stride, &check_norm),__FILE__,__LINE__);	

	printf("\n this is the norm: %E \n", check_norm);


	// destroy hiprand handle
	cuchebCheckError(hiprandDestroyGenerator(curand_gen),__FILE__,__LINE__);

	// destroy cublas handle
	cuchebCheckError(hipblasDestroy(handle),__FILE__,__LINE__);
	
	// copy device memory to host
	cuchebCheckError(hipMemcpy(x,dx,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(y,dy,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);

	// print values in host memory
	//printf("\nx and y after being set:\n\n");
	//for(int ii=0;ii<n;ii++){
	//	printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	//}
	//printf("\n");
	
	// free device memory
	cuchebCheckError(hipFree(dx),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dy),__FILE__,__LINE__);
	cuchebCheckError(hipFree(A),__FILE__,__LINE__);
	
	// free host memory
	delete[] x;
	delete[] y;
	
	// return
	return 0;

}
