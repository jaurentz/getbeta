#include <stdio.h>
#include <stdlib.h>
#include <cucheb.h>
#include <omp.h>


int main(void){

	// compute variables
	int n;
	double *x, *y, *dx, *dy;
	
	// set size
	n = 10;
	
	// allocate host memory
	x = new double[n];
	y = new double[n];
	
	// print values in host memory
	printf("\nx and y before being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// allocate device memory
	cuchebCheckError(hipMalloc(&dx,n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dy,n*sizeof(double)),__FILE__,__LINE__);
	
	// set device memory using CUCHEB
	cuchebCheckError(cuchebDinit(n,dx,1,1.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n,dy,1,0.0),__FILE__,__LINE__);
	
	// copy device memory to host
	cuchebCheckError(hipMemcpy(x,dx,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(y,dy,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print values in host memory
	printf("\nx and y after being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// free device memory
	cuchebCheckError(hipFree(dx),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dy),__FILE__,__LINE__);
	
	// free host memory
	delete[] x;
	delete[] y;
	
	// return
	return 0;

}
