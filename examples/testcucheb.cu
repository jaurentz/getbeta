#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cucheb.h>
#include <omp.h>


int main(void){

	// compute variables
	int n;
	double *x, *y, *dx, *dy;
	
	// set size
	n = 10;
	
	// allocate host memory
	x = new double[n];
	y = new double[n];
	
	// print values in host memory
	printf("\nx and y before being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// allocate device memory
	cuchebCheckError(hipMalloc(&dx,n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dy,n*sizeof(double)),__FILE__,__LINE__);
	
	// set device memory using CUCHEB
	cuchebCheckError(cuchebDinit(n,dx,1,1.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n,dy,1,0.0),__FILE__,__LINE__);

	// create cublas handle
	hipblasHandle_t handle;
	cuchebCheckError(hipblasCreate(&handle),__FILE__,__LINE__);
	
	// set pointer mode to host
	cuchebCheckError(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);

	// compute norm of dx
	double nrmx;
	cuchebCheckError(hipblasDnrm2(handle, n, dx, 1, &nrmx),__FILE__,__LINE__);

	// print norm of x
	printf("\nnorm of x = %+1.15e\n\n",nrmx);

	// compute norm of dy
	double nrmy;
	cuchebCheckError(hipblasDnrm2(handle, n, dy, 1, &nrmy),__FILE__,__LINE__);

	// print norm of y
	printf("\nnorm of y = %+1.15e\n\n",nrmy);

	// destroy cublas handle
	cuchebCheckError(hipblasDestroy(handle),__FILE__,__LINE__);
	
	// copy device memory to host
	cuchebCheckError(hipMemcpy(x,dx,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(y,dy,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print values in host memory
	printf("\nx and y after being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// free device memory
	cuchebCheckError(hipFree(dx),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dy),__FILE__,__LINE__);
	
	// free host memory
	delete[] x;
	delete[] y;
	
	// return
	return 0;

}
