#include "hip/hip_runtime.h"
#include <omp.h>
#include <hamop.h>
#include <cucheb.h>
extern "C" {
    #include "getbeta.h"
}


int main(void){

	//defining variables
	int n;
	double *x, *y, *dx, *dy, *A;
	
	// set size
	n = 5;
	
	// allocate host memory
	x = new double[n];
	y = new double[n];
	
	// print values in host memory
	printf("\nx and y before being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// allocate device memory
	cuchebCheckError(hipMalloc(&dx,n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dy,n*sizeof(double)),__FILE__,__LINE__);

	cuchebCheckError(hipMalloc(&A,n*n*sizeof(double)),__FILE__,__LINE__); 		

	// initialize hiprand
	hiprandGenerator_t curand_gen;
	cuchebCheckError(hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT),__FILE__,__LINE__);
	cuchebCheckError(hiprandSetPseudoRandomGeneratorSeed(curand_gen,time(NULL)),__FILE__,__LINE__);
	
	// random starting vector
	cuchebCheckError(hiprandGenerateNormalDouble(curand_gen,A,n*n,0.0,1.0),__FILE__,__LINE__);

	// set device memory using CUCHEB
	cuchebCheckError(cuchebDinit(n,dx,1,1.0),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(n,dy,1,1.0),__FILE__,__LINE__);

	//cuchebCheckError(cuchebDinit(n*n,A,1,cuRAND),__FILE__,__LINE__);
	//cuchebCheckError(cuchebDinit(n,A,n+1,1.0),__FILE__,__LINE__);
	

	// create cublas handle
	hipblasHandle_t handle;
	cuchebCheckError(hipblasCreate(&handle),__FILE__,__LINE__);
	
	// set pointer mode to host
	cuchebCheckError(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);

	// compute norm of dx
	//double nrmx;
	//cuchebCheckError(hipblasDnrm2(handle, n, dx, 1, &nrmx),__FILE__,__LINE__);

	// compute sum of dx
	//double sum;
	//cuchebCheckError(hipblasDasum(handle, n, dx, 1, &sum),__FILE__,__LINE__);

	// print sum of x
	//printf("\nthe sum of x = %E \n\n", sum);


	//computing alpha times a vector(dx) and storing it in (dy)
	//double alpha = 2.0;
	//int incx = 1; 
	//cuchebCheckError( hipblasDaxpy(handle, n, &alpha, dx, incx, dy, incx),__FILE__,__LINE__);


	//compute the dot product between the two vectors 
	//double z;
	//int incx = 1;
	//cuchebCheckError(hipblasDdot (handle, n, dx, incx, dy, incx, &z),__FILE__,__LINE__);

	//print the dot product "result"
	//printf("\n this is the dot product between x and y: \n %E", z); 


	//function performs matrix-vector multiplication y = alpha * OP(A) * x + beta * y
	int stride = 1;	
	//int n = 10;	
	double alpha = 2.0;
	double beta = 3.0;
	cuchebCheckError(hipblasDgemv(handle, HIPBLAS_OP_T, n, n, &alpha, A, n, dx, stride, &beta, dy, stride),__FILE__,__LINE__);	
	
	// compute norm of dy
	//double nrmy;
	//cuchebCheckError(hipblasDnrm2(handle, n, dy, 1, &nrmy),__FILE__,__LINE__);

	// print norm of y
	//printf("\nnorm of y = %+1.15e\n\n",nrmy);

	// destroy hiprand handle
	cuchebCheckError(hiprandDestroyGenerator(curand_gen),__FILE__,__LINE__);

	// destroy cublas handle
	cuchebCheckError(hipblasDestroy(handle),__FILE__,__LINE__);
	
	// copy device memory to host
	cuchebCheckError(hipMemcpy(x,dx,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(y,dy,n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);

	// print values in host memory
	printf("\nx and y after being set:\n\n");
	for(int ii=0;ii<n;ii++){
		printf("x[%d] = %+1.15e, y[%d] = %+1.15e\n",ii,x[ii],ii,y[ii]);
	}
	printf("\n");
	
	// free device memory
	cuchebCheckError(hipFree(dx),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dy),__FILE__,__LINE__);
	cuchebCheckError(hipFree(A),__FILE__,__LINE__);
	
	// free host memory
	delete[] x;
	delete[] y;
	
	// return
	return 0;

}
