#include "hip/hip_runtime.h"
#include <omp.h>
#include <hamop.h>
#include <cucheb.h>
extern "C" {
    #include "getbeta.h"
}


// spectrans
cuchebStatus_t spectrans(int n, const double *in, int incin, double *out, int incout, void* userdata){

	// set shape parameter
	double *tau = (double*)userdata;
	
	// compute function values
	for(int ii=0;ii<n;ii++){
		out[ii*incout] = exp(-(*tau)*in[ii*incin]*in[ii*incin]);
	}
	
	// return
	return CUCHEB_STATUS_SUCCESS;
}

// hamiltonian
void op(void* x, void* y, void* userdata){
	Hamop* HO = (Hamop*)userdata;
	HO->Mult((double*)x, (double*)y);
}


// driver
int main(){

	// declare hamiltonian
	Hamop HO;
	HO.setDims(3);
	HO.setNx(224);
	HO.setNy(224);
	HO.setNz(224);
	HO.setXminXmax(-10.0,10.0);
	HO.setYminYmax(-10.0,10.0);
	HO.setZminZmax(-10.0,10.0);
	HO.setXpotYpotZpot(1.0,2.0,3.0);
	
	// lanczos handle
	cuchebLanczosHandle LH;
	LH.n = HO.getNx()*HO.getNy()*HO.getNz();
	LH.numeigs = 2;
	LH.runlength = 60;
	LH.restarts = 1;
	
	// set n
	printf("\nn = %d\n",LH.n);
	
	// allocate memory for eigenvectors
	double *eigvecs;
	cuchebCheckError(hipMalloc(&eigvecs,LH.numeigs*LH.n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(cuchebDinit(LH.n,eigvecs,1,1.0),__FILE__,__LINE__);

	// allocate memory for residuals
	double *res, *ray;
	cuchebCheckError((void*)(ray = (double*)malloc(LH.numeigs*sizeof(double))),__FILE__,__LINE__);
	cuchebCheckError((void*)(res = (double*)malloc(LH.numeigs*sizeof(double))),__FILE__,__LINE__);
	
	// chebpoly memory
	double tau;
	double temp;
	double tol = 1e-2;
	double a, b;
	a = 0.0;
	ChebPoly SpecTrans;
	
	// chebop memory
	ChebOp SpecOp;

	// timer variables
	double begin, end;
	
	// specrad variable
	double specrad;
	
	// electric field points
	int numpoints = pow(2,3)+1;
	double chebpoints[numpoints];
	cheb_array(numpoints, &chebpoints[0], 0.1, -0.1);
	
	// polarization array
	double polarvals[numpoints];

	// begin timer
	begin = omp_get_wtime();
	
	// loop to compute polarization
	for(int jj=0;jj<numpoints;jj++){
	    // set e_field strength
	    HO.setXfieldYfieldZfield(chebpoints[jj],0.0,0.0);
	    //HO.print();
	
	
	    // compute specrad
	    cuchebCheckError(cuchebDspecrad(LH.n,op,(void*)&HO,&specrad),__FILE__,__LINE__);
	    b = specrad;
	
	    // end timer
	    //end = omp_get_wtime();
	    //printf("\nTime to compute specrad: %f (secs)\n",end-begin);
	    //printf("specrad = %e\n",specrad);
	
	    // chebpoly
	    tau = 1e5;
	    temp = tau/specrad/specrad;
	    SpecTrans = ChebPoly(&spectrans,(void*)&temp,&a,&b,&tol);
		
	    // set chebop
	    SpecOp = ChebOp(LH.n,op,(void*)&HO,&SpecTrans);
	    //SpecOp.print();
	
	    // begin timer
	    //begin = omp_get_wtime();
	
	    // call IRLM
	    LH.tol = 1e-14;
	    LH.numconv = 0;
	    LH.numrestarts = 0;
	    LH.nummatvecs = 0;
	    if(jj > 0){LH.runlength = 15;}	
	    cuchebCheckError(cuchebDeigs(&LH, &SpecOp, eigvecs),__FILE__,__LINE__);
	
	    // print Lanczos stats
	    printf("\nnumconv = %d\n",LH.numconv);
	    printf("numrestarts = %d\n",LH.numrestarts);
	    //printf("nummatvecs = %d\n\n",LH.nummatvecs);	
	
	    // compute rayleigh quotients and residuals
	    cuchebCheckError(rayleigh(LH.n,op,(void*)&HO,LH.numeigs,ray,eigvecs,res),__FILE__,__LINE__);
		
	    // print rayleigh quotients
	    //printf("\n");
	    for(int ii=0;ii<LH.numeigs;ii++){		
		    printf("rayleigh[%d] = %+1.15e, res[%d] = %+1.2e\n",ii,ray[ii],ii,res[ii]/specrad);
	    }
	    printf("\n");
	
	    // compute polarization
	    HO.Polarization(1.0, 0.0, 0.0, eigvecs, &polarvals[jj]);
	    	
	}
    
	// end timer
	end = omp_get_wtime();
	printf("\nTime to run IRLM: %f (secs)\n\n",end-begin);
	

    // print polarvals
    printf("\n");
    for(int ii=0;ii<numpoints;ii++){		
	    printf("(%+1.15e,%+1.15e)\n",chebpoints[ii],polarvals[ii]);
    }
    printf("\n");
    
    // compute beta
    double beta_abs;
    beta_abs = beta(numpoints, &chebpoints[0], &polarvals[0]);
    printf("beta_abs = %+1.15e\n\n",beta_abs);
	
	//free memory
	cuchebCheckError(hipFree(eigvecs),__FILE__,__LINE__);
	free(ray);
	free(res);

	// return
	return 0;

}

